#include <stdio.h> 
#include <hip/hip_runtime.h> 
#include <stdlib.h> 
#include <time.h>

//matrix multiplication done by the device
__global__ void DevMM(int *d_a,int *d_b, int *d_c, int N) { 
	
 	int row = blockIdx.y * blockDim.y + threadIdx.y; 
 	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;
	if( col < N && row < N) {
		for(int i = 0; i < N; i++) {
			sum += d_a[row * N + i] * d_b[i * N + col];
		}
		d_c[row * N + col] = sum;
	}
} 

//the matrix multiplcation done by the host
void HostMM(int * h_a, int* h_b, int* h_c, int N) {
	int row, col, k, sum;

	for(row = 0; row < N; row++) {
		for(col = 0; col < N; col++) {
			sum = 0;
			for(k = 0; k < N; k++) {
				sum += h_a[row * N + k] * h_b[k * N + col];
			}
			h_c[row * N + col] = sum;
		}
	}
}
//gets the usr intputs for the size of the array, number of threads and number of blocks
int getN(int* N) {
	printf("Please input size of array(NxN): ");
	scanf("%d", N);
	return 0;
}
int getB(int* B) {
	printf("Please input number of blocks(B): ");
	scanf("%d", B);
	return 0;
}
int getT(int* T) {
	printf("Please input number of threads(T): ");
	scanf("%d", T);
	return 0;
}

int main(void) { 
	
	int N, B, T;				
	int *a, *b, *c, *d; 			
	int *dev_a, *dev_b, *dev_c; 
	int size;				
	hipEvent_t start, stop;	// Time measuring events
	float etD, etH;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Get array dimensions from the user
	getN(&N);
	getB(&B);
	getT(&T);

	// Grid and block structures
	dim3 Grid(B, B); 	//amount blocks per grid in the x and y directions
	dim3 Block(T, T); 	//amount of threads per block in x and y idrections

	// Total size of arrays in bytes
	size = N * N * sizeof(int);

	// Set size of arrays
	a = (int*) malloc(size);
	b = (int*) malloc(size);
	c = (int*) malloc(size);
	d = (int*) malloc(size);

	// Initialize matrices with random numbers and intalize c and d to zero
	srand(1);
	for (int i = 0; i < N; i++) { //i is the row j is column
		for(int j = 0 ; j < N; j++) {
			
			int valA = rand() % 10;
			a[i * N + j] = valA;
			int valB = rand() % 10;
			b[i * N + j] = valB;
			c[i * N + j] = 0;
			d[i * N + j] = 0;
		}
	} 
	
	// Allocate memory on the device
	hipMalloc((void**)&dev_a, N * N * sizeof(int)); 
	hipMalloc((void**)&dev_b, N * N * sizeof(int)); 
	hipMalloc((void**)&dev_c, N * N * sizeof(int)); 

	// Copy the array from the host to the device
	hipMemcpy(dev_a, a , N * N * sizeof(int),hipMemcpyHostToDevice); 
	hipMemcpy(dev_b, b , N * N * sizeof(int),hipMemcpyHostToDevice); 
	hipMemcpy(dev_c, c , N * N * sizeof(int),hipMemcpyHostToDevice); 
	// Start timing the cuda computation
	hipEventRecord(start, 0);

	DevMM<<<Grid, Block>>>(dev_a, dev_b, dev_c, N);
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	// Measure end of compuatation
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&etD, start, stop);

	printf("Time spent by Device: %f ms.\n", etD);

	// Host computation
	hipEventRecord(start, 0);

	HostMM(a, b, d, N);
	
	// Measure end of compuatation
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&etH, start, stop);
	printf("Time spent by Host: %f ms.\n", etH);
	printf("Speedup Factor: %f\n",(etH/etD));
	//check if matrices are the same
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			if(c[i * N + j] != d[i * N + j]){
				printf("Matrices are not the same\n");
				exit(0);
			}	
		}
	}
	//}
	free(a); 
	free(b); 
	free(c); 
	free(d);
	hipFree(dev_a); 
	hipFree(dev_b); 
	hipFree(dev_c); 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0; 
}
