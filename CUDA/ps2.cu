#include <stdio.h> 
#include <hip/hip_runtime.h> 
#include <stdlib.h> 
#include <time.h>

int getN(int* N) {
	printf("Please input size of array(NxN): ");
	scanf("%d", N);
	return 0;
}
int getB(int* B) {
	printf("Please input number of blocks(B): ");
	scanf("%d", B);
	return 0;
}
int getT(int* T) {
	printf("Please input number of threads(T): ");
	scanf("%d", T);
	return 0;
}
__global__ void devSum(int *output, int *input, int n) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;

    	if (blockIdx.x > 0)
        	output[i] += input[blockIdx.x - 1];
} 

int main(void){
	int N, B, T;
	//get user input for arraysize, number of blocks, and number of threads
	getN(&N);
	getB(&B);
	getT(&T);

	int *a, *b, *c, *dev_a, *dev_b;
	
	// Time measuring events
	hipEvent_t start, stop;
	float etD, etH;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//Allocate memory on Device and Host
	a = (int*) malloc( sizeof(int) * N );
	b = (int*) malloc( sizeof(int) * N );
	c = (int*) malloc( sizeof(int) * N );
	hipMalloc((void**)&dev_a,N * sizeof(int));
	hipMalloc((void**)&dev_b,N * sizeof(int));

	srand(1);     //initialize random number generator 
	for (int i=0; i < N; i++){           // load array with some numbers 
		a[i] = rand() % 10; 
		//printf("%d ", a[i]);
		b[i] = 0;
	}
	//printf("\n");

	hipEventRecord(start, 0); 	//Start Host time
	int sum =0;
	for(int i = 0; i < N; i++){
		//printf("%d ",a[i]);
		sum += a[i];
		b[i] = sum;
		
	}
	hipEventRecord(stop, 0);	//End Host time and compute length of time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&etH, start, stop);

	//Copy input array with random Integers and a Zero-ed array used for output
	hipMemcpy(dev_a, a ,N * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b ,N * sizeof(int),hipMemcpyHostToDevice);

	// Start timing the Device computation
	hipEventRecord(start, 0);

	devSum<<<B, T>>>(dev_b, dev_a, N);

	//Copy output array from Device back to the Host
	hipMemcpy(c, dev_b, N * sizeof(int), hipMemcpyDeviceToHost);
	
	// Measure end of compuatation
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&etD, start, stop);

	//printf("\n");
	for(int i = 0; i < N; i++) {
			//printf("Dev: %d",c[i]);
			//printf(" host: %d\n",b[i]);

			if(c[i] != b[i]){//Check if Device and Host are different, if they are close the program
				printf("Results are not the same\n");
				exit(0);
			}	
		}
	printf("\nHost and Device produce the same results.");
	printf("\n");
	printf("Time spent by Device: %f ms.\n", etD);
	printf("Time spent by Host: %f ms.\n", etH);
	printf("Speedup Factor: %f\n",(etH/etD));
	hipFree(dev_a);
	hipFree(dev_b);
	
return 0;
}
